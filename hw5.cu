#include "hip/hip_runtime.h"
// #define DEBUG

#include <nppdefs.h>
#include <omp.h>
#include <pthread.h>
#include <sm_60_atomic_functions.h>

#include <cmath>
#include <fstream>
#include <iomanip>
#include <limits>
#include <set>
#include <stdexcept>
#include <string>
#include <vector>

#ifdef DEBUG
#include <chrono>
#define __debug_printf(fmt, args...) printf(fmt, ##args);
#define __START_TIME(ID) auto start_##ID = std::chrono::high_resolution_clock::now();
#define __END_TIME(ID)                                                                                         \
    auto stop_##ID = std::chrono::high_resolution_clock::now();                                                \
    int duration_##ID = std::chrono::duration_cast<std::chrono::milliseconds>(stop_##ID - start_##ID).count(); \
    __debug_printf("duration of %s: %d milliseconds\n", #ID, duration_##ID);
#define CUDA_CALL(F)                                                          \
    if ((F != hipSuccess)) {                                                 \
        printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
               __FILE__, __LINE__);                                           \
        exit(-1);                                                             \
    }
#define CUDA_CHECK()                                                          \
    if ((hipPeekAtLastError()) != hipSuccess) {                             \
        printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
               __FILE__, __LINE__ - 1);                                       \
        exit(-1);                                                             \
    }
#else
#define __debug_printf(fmt, args...)
#define __START_TIME(ID)
#define __END_TIME(ID)
#define CUDA_CALL(F) (F)
#define CUDA_CHECK()
#endif

namespace param {
const int n_steps = 200000;
const int n_sync_steps = 4000;
const double dt = 60;
const double eps = 1e-3;
const double sq_eps = eps * eps;
const double G = 6.674e-11;
double gravity_device_mass(double m0, double t) {
    return m0 + 0.5 * m0 * fabs(sin(t / 6000));
}
__device__ double gravity_device_mass_gpu(double m0, double t) {
    return m0 + 0.5 * m0 * fabs(sin(t / 6000));
}
const double planet_radius = 1e7;
const double sq_planet_radius = planet_radius * planet_radius;
const double missile_speed = 1e6;
double get_missile_dist(int step) { return (missile_speed * missile_speed * dt * dt) * (step * step); }
double get_missile_cost(double t) { return 1e5 + 1e3 * t; }
__device__ double get_missile_cost_gpu(double t) { return 1e5 + 1e3 * t; }

const int threads_per_block = 512;
const int cuda_nstreams = 3;
}  // namespace param

void read_input(const char* filename, int& n, double*& qx, double*& qy, double*& qz,
                double*& vx, double*& vy, double*& vz, double*& m, int& device_cnt, int*& device_id) {
    std::ifstream fin(filename);
    int planet, asteroid;
    fin >> n >> planet >> asteroid;

    std::string type;
    std::vector<double> tmp_qx(n);
    std::vector<double> tmp_qy(n);
    std::vector<double> tmp_qz(n);
    std::vector<double> tmp_vx(n);
    std::vector<double> tmp_vy(n);
    std::vector<double> tmp_vz(n);
    std::vector<double> tmp_m(n);
    std::vector<int> tmp_devices;
    std::set<int> indices;
    for (int i = 0; i < n; i++) {
        indices.insert(i);
        fin >> tmp_qx[i] >> tmp_qy[i] >> tmp_qz[i] >> tmp_vx[i] >> tmp_vy[i] >> tmp_vz[i] >> tmp_m[i] >> type;
        if (type == "device") {
            tmp_devices.push_back(i);
        }
    }

    qx = (double*)malloc(n * sizeof(double));
    qy = (double*)malloc(n * sizeof(double));
    qz = (double*)malloc(n * sizeof(double));
    vx = (double*)malloc(n * sizeof(double));
    vy = (double*)malloc(n * sizeof(double));
    vz = (double*)malloc(n * sizeof(double));
    m = (double*)malloc(n * sizeof(double));
    device_id = (int*)malloc(n * sizeof(int));
    device_cnt = tmp_devices.size();
    for (int i = 0; i < n; i++) {
        int tmp_i;
        if (i == 0) {
            tmp_i = planet;
        } else if (i == 1) {
            tmp_i = asteroid;
        } else if (i < device_cnt + 2) {
            tmp_i = tmp_devices[i - 2];
            device_id[i] = tmp_devices[i - 2];
        } else {
            tmp_i = *indices.begin();
        }
        qx[i] = tmp_qx[tmp_i];
        qy[i] = tmp_qy[tmp_i];
        qz[i] = tmp_qz[tmp_i];
        vx[i] = tmp_vx[tmp_i];
        vy[i] = tmp_vy[tmp_i];
        vz[i] = tmp_vz[tmp_i];
        m[i] = tmp_m[tmp_i];
        indices.erase(tmp_i);
    }
}

void write_output(const char* filename, double min_dist, int hit_time_step,
                  int gravity_device_id, double missile_cost) {
    std::ofstream fout(filename);
    fout << std::scientific
         << std::setprecision(std::numeric_limits<double>::digits10 + 1) << min_dist
         << '\n'
         << hit_time_step << '\n'
         << gravity_device_id << ' ' << missile_cost << '\n';
}

template <class T>
__global__ void clear_array_gpu(int n, T* array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        array[i] = (T)0;
    }
}

__global__ void set_isdevice_gpu(int device_cnt, int* devices, bool* isdevice) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < device_cnt) {
        isdevice[devices[i]] = true;
    }
}

__global__ void compute_accelerations_1_gpu(const int step, const int n, const double* qx, const double* qy, const double* qz, double* vx, double* vy, double* vz, double* ax, double* ay, double* az, const double* m, const int device_cnt) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i = index / n;
    int j = index % n;

    // compute accelerations
    if (i < n && j < n && i != j) {
        double mj = m[j];
        if (j > 1 && j < device_cnt + 2) {
            mj = 0;
        }
        double dx = qx[j] - qx[i];
        double dy = qy[j] - qy[i];
        double dz = qz[j] - qz[i];
        double dist3 =
            pow(dx * dx + dy * dy + dz * dz + param::eps * param::eps, 1.5);

        atomicAdd(&ax[i], param::G * mj * dx / dist3);
        atomicAdd(&ay[i], param::G * mj * dy / dist3);
        atomicAdd(&az[i], param::G * mj * dz / dist3);
    }
}

__global__ void compute_accelerations_2_gpu(const int step, const int n, const double* qx, const double* qy, const double* qz, double* vx, double* vy, double* vz, double* ax, double* ay, double* az, const double* m, const int device_cnt) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i = index / n;
    int j = index % n;

    // compute accelerations
    if (i < n && j < n && i != j) {
        double mj = m[j];
        if (j > 1 && j < device_cnt + 2) {
            mj = param::gravity_device_mass_gpu(mj, step * param::dt);
        }
        double dx = qx[j] - qx[i];
        double dy = qy[j] - qy[i];
        double dz = qz[j] - qz[i];
        double dist3 =
            pow(dx * dx + dy * dy + dz * dz + param::eps * param::eps, 1.5);

        atomicAdd(&ax[i], param::G * mj * dx / dist3);
        atomicAdd(&ay[i], param::G * mj * dy / dist3);
        atomicAdd(&az[i], param::G * mj * dz / dist3);
    }
}

__global__ void compute_accelerations_3_gpu(const bool* hit, const bool* destroyed, const int step, const int n, const int d, const double* qx, const double* qy, const double* qz, double* vx, double* vy, double* vz, double* ax, double* ay, double* az, const double* m, const int device_cnt) {
    if (*hit)
        return;
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i = index / n;
    int j = index % n;

    // compute accelerations
    if (i < n && j < n && i != j) {
        double mj = m[j];
        if (j > 1 && j < device_cnt + 2)
            mj = param::gravity_device_mass_gpu(mj, step * param::dt);

        double dx = qx[j] - qx[i];
        double dy = qy[j] - qy[i];
        double dz = qz[j] - qz[i];
        double dist3 =
            pow(dx * dx + dy * dy + dz * dz + param::eps * param::eps, 1.5);

        atomicAdd(&ax[i], param::G * mj * dx / dist3);
        atomicAdd(&ay[i], param::G * mj * dy / dist3);
        atomicAdd(&az[i], param::G * mj * dz / dist3);
    }
}

__global__ void update_velocities_gpu(const int n, double* vx, double* vy, double* vz, double* ax, double* ay, double* az) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // update velocities
    if (i < n) {
        vx[i] += ax[i] * param::dt;
        ax[i] = 0;
    } else if (i < 2 * n) {
        vy[i - n] += ay[i - n] * param::dt;
        ay[i - n] = 0;
    } else if (i < 3 * n) {
        vz[i - 2 * n] += az[i - 2 * n] * param::dt;
        az[i - 2 * n] = 0;
    }
}

__global__ void clear_a_gpu(const int n, double* ax, double* ay, double* az) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        ax[i] = 0;
    } else if (i < 2 * n) {
        ay[i - n] = 0;
    } else if (i < 3 * n) {
        az[i - 2 * n] = 0;
    }
}

__global__ void update_positions_gpu(const int n, double* qx, double* qy, double* qz, const double* vx, const double* vy, const double* vz, const double* ax, const double* ay, const double* az) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // update positions
    if (i < n) {
        qx[i] += (vx[i] + ax[i] * param::dt) * param::dt;
    } else if (i < 2 * n) {
        qy[i - n] += (vy[i - n] + ay[i - n] * param::dt) * param::dt;
    } else if (i < 3 * n) {
        qz[i - 2 * n] += (vz[i - 2 * n] + az[i - 2 * n] * param::dt) * param::dt;
    }
}

__global__ void problem1(double* min_dist, const int n, const double* qx, const double* qy, const double* qz) {
    double dx = qx[0] - qx[1];
    double dy = qy[0] - qy[1];
    double dz = qz[0] - qz[1];
    double tmp_dst = sqrt(dx * dx + dy * dy + dz * dz);
    if (tmp_dst < *min_dist)
        *min_dist = tmp_dst;
}

__global__ void problem2(int* hit_time_step, const int step, const int n, const double* qx, const double* qy, const double* qz) {
    if (*hit_time_step != -2)
        return;
    double dx = qx[0] - qx[1];
    double dy = qy[0] - qy[1];
    double dz = qz[0] - qz[1];
    if (dx * dx + dy * dy + dz * dz < param::planet_radius * param::planet_radius) {
        *hit_time_step = step;
    }
}

__global__ void problem3(bool* hit, bool* destroyed, double* cost, const int step, const int n, const int d, const double* qx, const double* qy, const double* qz, double* m) {
    if (*hit)
        return;
    double dx = qx[0] - qx[1];
    double dy = qy[0] - qy[1];
    double dz = qz[0] - qz[1];
    if (dx * dx + dy * dy + dz * dz < param::planet_radius * param::planet_radius) {
        *hit = true;
        return;
    }
    if (!*destroyed) {
        dx = qx[0] - qx[d];
        dy = qy[0] - qy[d];
        dz = qz[0] - qz[d];
        double missle_dist = (param::missile_speed * param::dt) * step;
        if (dx * dx + dy * dy + dz * dz < missle_dist * missle_dist) {
            *destroyed = true;
            *cost = param::get_missile_cost_gpu((step + 1) * param::dt);
            m[d] = 0;
        }
    }
}

int main(int argc, char** argv) {
    if (argc != 3) {
        throw std::runtime_error("must supply 2 arguments");
    }
    int n;
    double *qx0, *qy0, *qz0;
    double *vx0, *vy0, *vz0;
    double* m0;
    int device_cnt;
    int* device_id;
    read_input(argv[1], n, qx0, qy0, qz0, vx0, vy0, vz0, m0, device_cnt, device_id);

    double min_dist = std::numeric_limits<double>::infinity();
    int hit_time_step = -2;
    int gravity_device_id = -1;
    double missile_cost = 0;

    dim3 BlockDim(param::threads_per_block);
    // dim3 GridDim(ceil((float)n / param::threads_per_block));
    auto GridDim = [&](int n) -> dim3 {
        return (ceil((float)n / param::threads_per_block));
    };

#pragma omp parallel for num_threads(2)
    for (int task = 0; task < 2; task++) {
        CUDA_CALL(hipSetDevice(omp_get_thread_num()));
        hipStream_t streams[param::cuda_nstreams];
        for (int i = 0; i < param::cuda_nstreams; i++) {
            CUDA_CALL(hipStreamCreate(&streams[i]));
        }

        double *p_qx, *p_qy, *p_qz;
        double *g_qx, *g_qy, *g_qz;
        double *g_vx, *g_vy, *g_vz;
        double *g_ax, *g_ay, *g_az;
        double* g_m;
        double* g_min_dist;
        int* g_hit_time_step;

        CUDA_CALL(hipMalloc(&p_qx, 2 * sizeof(double)));
        CUDA_CALL(hipMalloc(&p_qy, 2 * sizeof(double)));
        CUDA_CALL(hipMalloc(&p_qz, 2 * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_qx, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_qy, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_qz, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_vx, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_vy, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_vz, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_ax, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_ay, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_az, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_m, n * sizeof(double)));
        if (task == 0) {
            CUDA_CALL(hipMallocManaged(&g_min_dist, sizeof(double)));
            CUDA_CALL(hipMemPrefetchAsync(g_min_dist, sizeof(double), hipCpuDeviceId, streams[1]));
            *g_min_dist = std::numeric_limits<double>::infinity();
            CUDA_CALL(hipMemPrefetchAsync(g_min_dist, sizeof(double), omp_get_thread_num(), streams[1]));
        } else if (task == 1) {
            CUDA_CALL(hipMallocManaged(&g_hit_time_step, sizeof(int)));
            CUDA_CALL(hipMemPrefetchAsync(g_hit_time_step, sizeof(int), hipCpuDeviceId, streams[1]));
            *g_hit_time_step = -2;
            CUDA_CALL(hipMemPrefetchAsync(g_hit_time_step, sizeof(int), omp_get_thread_num(), streams[1]));
        }

        CUDA_CALL(hipMemcpyAsync(p_qx, qx0, 2 * sizeof(double), hipMemcpyHostToDevice, streams[1]));
        CUDA_CALL(hipMemcpyAsync(p_qy, qy0, 2 * sizeof(double), hipMemcpyHostToDevice, streams[1]));
        CUDA_CALL(hipMemcpyAsync(p_qz, qz0, 2 * sizeof(double), hipMemcpyHostToDevice, streams[1]));
        CUDA_CALL(hipMemcpyAsync(g_qx, qx0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_qy, qy0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_qz, qz0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_vx, vx0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_vy, vy0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_vz, vz0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_m, m0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        clear_a_gpu<<<GridDim(3 * n), BlockDim, 0, streams[1]>>>(n, g_ax, g_ay, g_az);

        for (int step = 0; step <= param::n_steps; step++) {
            if (step > 0) {
                if (task == 0)
                    compute_accelerations_1_gpu<<<GridDim(n * n), BlockDim, 0, streams[0]>>>(step, n, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz, g_ax, g_ay, g_az, g_m, device_cnt);
                else
                    compute_accelerations_2_gpu<<<GridDim(n * n), BlockDim, 0, streams[0]>>>(step, n, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz, g_ax, g_ay, g_az, g_m, device_cnt);
                update_positions_gpu<<<GridDim(3 * n), BlockDim, 0, streams[0]>>>(n, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz, g_ax, g_ay, g_az);
                CUDA_CALL(hipStreamSynchronize(streams[1]));
                CUDA_CALL(hipMemcpy(p_qx, g_qx, 2 * sizeof(double), hipMemcpyDeviceToDevice));
                CUDA_CALL(hipMemcpy(p_qy, g_qy, 2 * sizeof(double), hipMemcpyDeviceToDevice));
                CUDA_CALL(hipMemcpy(p_qz, g_qz, 2 * sizeof(double), hipMemcpyDeviceToDevice));
                update_velocities_gpu<<<GridDim(3 * n), BlockDim, 0, streams[0]>>>(n, g_vx, g_vy, g_vz, g_ax, g_ay, g_az);
            }
            if (task == 0) {
                problem1<<<1, 1, 0, streams[1]>>>(g_min_dist, n, p_qx, p_qy, p_qz);
            } else if (task == 1) {
                problem2<<<1, 1, 0, streams[1]>>>(g_hit_time_step, step, n, p_qx, p_qy, p_qz);
                if (step % param::n_sync_steps == param::n_sync_steps - 1) {
                    CUDA_CALL(hipMemPrefetchAsync(g_hit_time_step, sizeof(int), hipCpuDeviceId, streams[1]));
                    CUDA_CALL(hipStreamSynchronize(streams[1]));
                    hit_time_step = *g_hit_time_step;
                    if (hit_time_step != -2)
                        break;
                }
            }
        }
        if (task == 0) {
            CUDA_CALL(hipMemPrefetchAsync(g_min_dist, sizeof(double), hipCpuDeviceId, streams[1]));
            CUDA_CALL(hipStreamSynchronize(streams[1]));
            min_dist = *g_min_dist;
        } else if (task == 1) {
            if (hit_time_step == -2) {
                CUDA_CALL(hipMemPrefetchAsync(g_hit_time_step, sizeof(int), hipCpuDeviceId, streams[1]));
                CUDA_CALL(hipStreamSynchronize(streams[1]));
                hit_time_step = *g_hit_time_step;
            }
        }
        for (int i = 0; i < param::cuda_nstreams; i++) {
            CUDA_CALL(hipStreamDestroy(streams[i]));
        }
        CUDA_CALL(hipFree(p_qx));
        CUDA_CALL(hipFree(p_qy));
        CUDA_CALL(hipFree(p_qz));
        CUDA_CALL(hipFree(g_qx));
        CUDA_CALL(hipFree(g_qy));
        CUDA_CALL(hipFree(g_qz));
        CUDA_CALL(hipFree(g_vx));
        CUDA_CALL(hipFree(g_vy));
        CUDA_CALL(hipFree(g_vz));
        CUDA_CALL(hipFree(g_ax));
        CUDA_CALL(hipFree(g_ay));
        CUDA_CALL(hipFree(g_az));
        CUDA_CALL(hipFree(g_m));
        if (task == 0)
            CUDA_CALL(hipFree(g_min_dist));
        else if (task == 1)
            CUDA_CALL(hipFree(g_hit_time_step));
    }  // omp end

    if (hit_time_step != -2) {
        // Problem 3
        gravity_device_id = -1;
        missile_cost = std::numeric_limits<double>::infinity();
#pragma omp parallel for schedule(static) num_threads(2)
        for (int di = 0; di < device_cnt; di++) {
            int thread_id = omp_get_thread_num();
            hipSetDevice(thread_id);
            hipStream_t streams[param::cuda_nstreams];
            for (int i = 0; i < param::cuda_nstreams; i++) {
                hipStreamCreate(&streams[i]);
            }

            int d = di + 2;
            double *p_qx, *p_qy, *p_qz;
            double *g_qx, *g_qy, *g_qz;
            double *g_vx, *g_vy, *g_vz;
            double *g_ax, *g_ay, *g_az;
            double* g_m;
            bool hit = false;
            bool* g_hit;
            bool* g_destroyed;
            double* g_cost;

            CUDA_CALL(hipMalloc(&p_qx, (2 + device_cnt) * sizeof(double)));
            CUDA_CALL(hipMalloc(&p_qy, (2 + device_cnt) * sizeof(double)));
            CUDA_CALL(hipMalloc(&p_qz, (2 + device_cnt) * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_qx, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_qy, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_qz, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_vx, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_vy, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_vz, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_ax, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_ay, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_az, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_m, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_destroyed, sizeof(bool)));

            CUDA_CALL(hipMallocManaged(&g_hit, sizeof(bool)));
            CUDA_CALL(hipMemPrefetchAsync(g_hit, sizeof(bool), hipCpuDeviceId, streams[1]));
            *g_hit = false;
            CUDA_CALL(hipMemPrefetchAsync(g_hit, sizeof(bool), thread_id, streams[1]));
            CUDA_CALL(hipMallocManaged(&g_cost, sizeof(double)));
            CUDA_CALL(hipMemPrefetchAsync(g_cost, sizeof(bool), hipCpuDeviceId, streams[1]));
            *g_cost = std::numeric_limits<double>::infinity();
            CUDA_CALL(hipMemPrefetchAsync(g_cost, sizeof(bool), thread_id, streams[1]));

            CUDA_CALL(hipMemcpyAsync(p_qx, qx0, (2 + device_cnt) * sizeof(double), hipMemcpyHostToDevice, streams[1]));
            CUDA_CALL(hipMemcpyAsync(p_qy, qy0, (2 + device_cnt) * sizeof(double), hipMemcpyHostToDevice, streams[1]));
            CUDA_CALL(hipMemcpyAsync(p_qz, qz0, (2 + device_cnt) * sizeof(double), hipMemcpyHostToDevice, streams[1]));
            CUDA_CALL(hipMemcpyAsync(g_qx, qx0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_qy, qy0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_qz, qz0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_vx, vx0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_vy, vy0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_vz, vz0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_m, m0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            clear_a_gpu<<<GridDim(3 * n), BlockDim, 0, streams[0]>>>(n, g_ax, g_ay, g_az);
            CUDA_CALL(hipMemsetAsync(g_destroyed, m0[d] == 0, sizeof(bool), streams[1]));

            // hit break
            for (int step = 0; step <= param::n_steps && !hit; step++) {
                if (step > 0) {
                    hipStreamSynchronize(streams[1]);
                    compute_accelerations_3_gpu<<<GridDim(n * n), BlockDim, 0, streams[0]>>>(g_hit, g_destroyed, step, n, d, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz, g_ax, g_ay, g_az, g_m, device_cnt);
                    update_positions_gpu<<<GridDim(3 * n), BlockDim, 0, streams[0]>>>(n, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz, g_ax, g_ay, g_az);
                    CUDA_CALL(hipMemcpy(p_qx, g_qx, (2 + device_cnt) * sizeof(double), hipMemcpyDeviceToDevice));
                    CUDA_CALL(hipMemcpy(p_qy, g_qy, (2 + device_cnt) * sizeof(double), hipMemcpyDeviceToDevice));
                    CUDA_CALL(hipMemcpy(p_qz, g_qz, (2 + device_cnt) * sizeof(double), hipMemcpyDeviceToDevice));
                    update_velocities_gpu<<<GridDim(3 * n), BlockDim, 0, streams[0]>>>(n, g_vx, g_vy, g_vz, g_ax, g_ay, g_az);
                }
                problem3<<<1, 1, 0, streams[1]>>>(g_hit, g_destroyed, g_cost, step, n, d, p_qx, p_qy, p_qz, g_m);
                if (step % param::n_sync_steps == param::n_sync_steps - 1) {
                    CUDA_CALL(hipMemPrefetchAsync(g_hit, sizeof(bool), hipCpuDeviceId, streams[1]));
                    hipStreamSynchronize(streams[1]);
                    hit = *g_hit;
                    if (hit)
                        break;
                }
            }
            if (!hit) {
                CUDA_CALL(hipMemPrefetchAsync(g_hit, sizeof(bool), hipCpuDeviceId, streams[1]));
                CUDA_CALL(hipMemPrefetchAsync(g_cost, sizeof(double), hipCpuDeviceId, streams[1]));
                hipStreamSynchronize(streams[1]);
#pragma omp critical
                if (!*g_hit && *g_cost < missile_cost) {
                    gravity_device_id = d;
                    missile_cost = *g_cost;
                }
            }
            for (int i = 0; i < param::cuda_nstreams; i++) {
                hipStreamDestroy(streams[i]);
            }
            CUDA_CALL(hipFree(p_qx));
            CUDA_CALL(hipFree(p_qy));
            CUDA_CALL(hipFree(p_qz));
            CUDA_CALL(hipFree(g_qx));
            CUDA_CALL(hipFree(g_qy));
            CUDA_CALL(hipFree(g_qz));
            CUDA_CALL(hipFree(g_vx));
            CUDA_CALL(hipFree(g_vy));
            CUDA_CALL(hipFree(g_vz));
            CUDA_CALL(hipFree(g_ax));
            CUDA_CALL(hipFree(g_ay));
            CUDA_CALL(hipFree(g_az));
            CUDA_CALL(hipFree(g_m));
            CUDA_CALL(hipFree(g_hit));
            CUDA_CALL(hipFree(g_destroyed));
            CUDA_CALL(hipFree(g_cost));
        }  // omp end
        if (gravity_device_id == -1) {
            missile_cost = 0;
        } else {
            gravity_device_id = device_id[gravity_device_id];
        }
    }

    write_output(argv[2], min_dist, hit_time_step, gravity_device_id, missile_cost);

    CUDA_CHECK();

    free(qx0);
    free(qy0);
    free(qz0);
    free(vx0);
    free(vy0);
    free(vz0);
    free(m0);
    free(device_id);
}

/*
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b20.in outputs/b20.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b30.in outputs/b30.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b40.in outputs/b40.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b50.in outputs/b50.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b60.in outputs/b60.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b70.in outputs/b70.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b80.in outputs/b80.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b90.in outputs/b90.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b100.in outputs/b100.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b200.in outputs/b200.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b512.in outputs/b512.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b1024.in outputs/b1024.out

make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b20.in outputs/b20.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b30.in outputs/b30.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b40.in outputs/b40.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b50.in outputs/b50.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b60.in outputs/b60.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b70.in outputs/b70.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b80.in outputs/b80.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b90.in outputs/b90.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b100.in outputs/b100.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b200.in outputs/b200.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b512.in outputs/b512.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b1024.in outputs/b1024.out

make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b20.in outputs/b20.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b30.in outputs/b30.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b40.in outputs/b40.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b50.in outputs/b50.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b60.in outputs/b60.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b70.in outputs/b70.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b80.in outputs/b80.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b90.in outputs/b90.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b100.in outputs/b100.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b200.in outputs/b200.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b512.in outputs/b512.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b1024.in outputs/b1024.out
 */