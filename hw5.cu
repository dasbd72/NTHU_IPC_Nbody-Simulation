#include "hip/hip_runtime.h"
#define DEBUG

#include <nppdefs.h>
#include <omp.h>
#include <pthread.h>
#include <sm_60_atomic_functions.h>

#include <cmath>
#include <fstream>
#include <iomanip>
#include <limits>
#include <stdexcept>
#include <string>
#include <vector>

#ifdef DEBUG
#include <chrono>
#define __debug_printf(fmt, args...) printf(fmt, ##args);
#define __START_TIME(ID) auto start_##ID = std::chrono::high_resolution_clock::now();
#define __END_TIME(ID)                                                                                         \
    auto stop_##ID = std::chrono::high_resolution_clock::now();                                                \
    int duration_##ID = std::chrono::duration_cast<std::chrono::milliseconds>(stop_##ID - start_##ID).count(); \
    __debug_printf("duration of %s: %d milliseconds\n", #ID, duration_##ID);
#define CUDA_CALL(F)                                                          \
    if ((F != hipSuccess)) {                                                 \
        printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
               __FILE__, __LINE__);                                           \
        exit(-1);                                                             \
    }
#define CUDA_CHECK()                                                          \
    if ((hipPeekAtLastError()) != hipSuccess) {                             \
        printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
               __FILE__, __LINE__ - 1);                                       \
        exit(-1);                                                             \
    }
#else
#define __debug_printf(fmt, args...)
#define __START_TIME(ID)
#define __END_TIME(ID)
#define CUDA_CALL(F) (F)
#define CUDA_CHECK()
#endif

namespace param {
const int n_steps = 200000;
const double dt = 60;
const double eps = 1e-3;
const double sq_eps = eps * eps;
const double G = 6.674e-11;
double gravity_device_mass(double m0, double t) {
    return m0 + 0.5 * m0 * fabs(sin(t / 6000));
}
__device__ double gravity_device_mass_gpu(double m0, double t) {
    return m0 + 0.5 * m0 * fabs(sin(t / 6000));
}
const double planet_radius = 1e7;
const double sq_planet_radius = planet_radius * planet_radius;
const double missile_speed = 1e6;
double get_missile_cost(double t) { return 1e5 + 1e3 * t; }

const int threads_per_block = 256;
const int cuda_nstreams = 3;
}  // namespace param

void read_input(const char* filename, int& n, int& planet, int& asteroid, double*& qx, double*& qy, double*& qz,
                double*& vx, double*& vy, double*& vz, double*& m, int& device_cnt, int*& devices) {
    std::ifstream fin(filename);
    std::string type;
    std::vector<int> tmp_devices;
    fin >> n >> planet >> asteroid;
    qx = (double*)malloc(n * sizeof(double));
    qy = (double*)malloc(n * sizeof(double));
    qz = (double*)malloc(n * sizeof(double));
    vx = (double*)malloc(n * sizeof(double));
    vy = (double*)malloc(n * sizeof(double));
    vz = (double*)malloc(n * sizeof(double));
    m = (double*)malloc(n * sizeof(double));
    for (int i = 0; i < n; i++) {
        fin >> qx[i] >> qy[i] >> qz[i] >> vx[i] >> vy[i] >> vz[i] >> m[i] >> type;
        if (type == "device") {
            tmp_devices.push_back(i);
        }
    }
    device_cnt = tmp_devices.size();
    devices = (int*)malloc(device_cnt * sizeof(int));
    for (int i = 0; i < device_cnt; i++) {
        devices[i] = tmp_devices[i];
    }
}

void write_output(const char* filename, double min_dist, int hit_time_step,
                  int gravity_device_id, double missile_cost) {
    std::ofstream fout(filename);
    fout << std::scientific
         << std::setprecision(std::numeric_limits<double>::digits10 + 1) << min_dist
         << '\n'
         << hit_time_step << '\n'
         << gravity_device_id << ' ' << missile_cost << '\n';
}

template <class T>
__global__ void clear_array_gpu(int n, T* array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        array[i] = (T)0;
    }
}

__global__ void set_isdevice_gpu(int device_cnt, int* devices, bool* isdevice) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < device_cnt) {
        isdevice[devices[i]] = true;
    }
}

__global__ void compute_accelerations_gpu(const bool isProblem1, const int step, const int n, const double* qx, const double* qy, const double* qz, const double* vx, const double* vy, const double* vz, double* ax, double* ay, double* az, const double* m, const bool* isdevice) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int i = index / n;
    int j = index % n;

    // compute accelerations
    if (i < n && j < n && i != j) {
        double mj = m[j];
        if (isdevice[j]) {
            if (isProblem1)
                mj = 0;
            else
                mj = param::gravity_device_mass_gpu(mj, step * param::dt);
        }
        double dx = qx[j] - qx[i];
        double dy = qy[j] - qy[i];
        double dz = qz[j] - qz[i];
        double dist3 =
            pow(dx * dx + dy * dy + dz * dz + param::eps * param::eps, 1.5);

        atomicAdd(&ax[i], param::G * mj * dx / dist3);
        atomicAdd(&ay[i], param::G * mj * dy / dist3);
        atomicAdd(&az[i], param::G * mj * dz / dist3);
    }
}

__global__ void update_velocities_gpu(int n, double* vx, double* vy, double* vz, double* ax, double* ay, double* az) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // update velocities
    if (i < n) {
        vx[i] += ax[i] * param::dt;
    } else if (i < 2 * n) {
        vy[i - n] += ay[i - n] * param::dt;
    } else if (i < 3 * n) {
        vz[i - 2 * n] += az[i - 2 * n] * param::dt;
    }
}

__global__ void clear_a_gpu(int n, double* ax, double* ay, double* az) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        ax[i] = 0;
    } else if (i < 2 * n) {
        ay[i - n] = 0;
    } else if (i < 3 * n) {
        az[i - 2 * n] = 0;
    }
}

__global__ void update_positions_gpu(int n, double* qx, double* qy, double* qz, double* vx, double* vy, double* vz) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // update positions
    if (i < n) {
        qx[i] += vx[i] * param::dt;
    } else if (i < 2 * n) {
        qy[i - n] += vy[i - n] * param::dt;
    } else if (i < 3 * n) {
        qz[i - 2 * n] += vz[i - 2 * n] * param::dt;
    }
}

int main(int argc, char** argv) {
    if (argc != 3) {
        throw std::runtime_error("must supply 2 arguments");
    }
    int n, planet, asteroid;
    double *qx0, *qy0, *qz0, *vx0, *vy0, *vz0, *m0;
    int device_cnt;
    int* devices;
    read_input(argv[1], n, planet, asteroid, qx0, qy0, qz0, vx0, vy0, vz0, m0, device_cnt, devices);

    double min_dist = std::numeric_limits<double>::infinity();
    int hit_time_step = -2;
    int gravity_device_id = -1;
    double missile_cost = 0;

    dim3 BlockDim(param::threads_per_block);
    // dim3 GridDim(ceil((float)n / param::threads_per_block));
    auto GridDim = [&](int n) -> dim3 {
        return (ceil((float)n / param::threads_per_block));
    };

#pragma omp parallel num_threads(2)
    {
        int thread_id = omp_get_thread_num();
        hipSetDevice(thread_id);
        hipStream_t streams[param::cuda_nstreams];
        for (int i = 0; i < param::cuda_nstreams; i++) {
            hipStreamCreate(&streams[i]);
        }

        double *qx, *qy, *qz;
        double *g_qx, *g_qy, *g_qz, *g_vx, *g_vy, *g_vz, *g_ax, *g_ay, *g_az, *g_m;
        bool* g_isdevice;
        int* g_devices;

        CUDA_CALL(hipMalloc(&g_qx, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_qy, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_qz, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_vx, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_vy, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_vz, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_ax, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_ay, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_az, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_m, n * sizeof(double)));
        CUDA_CALL(hipMalloc(&g_isdevice, n * sizeof(bool)));
        CUDA_CALL(hipMalloc(&g_devices, device_cnt * sizeof(int)));

        CUDA_CALL(hipMemcpyAsync(g_qx, qx0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_qy, qy0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_qz, qz0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_vx, vx0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_vy, vy0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_vz, vz0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_m, m0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
        CUDA_CALL(hipMemcpyAsync(g_devices, devices, device_cnt * sizeof(int), hipMemcpyHostToDevice, streams[0]));
        clear_a_gpu<<<GridDim(3 * n), BlockDim, 0, streams[1]>>>(n, g_ax, g_ay, g_az);
        clear_array_gpu<<<GridDim(n), BlockDim, 0, streams[0]>>>(n, g_isdevice);
        set_isdevice_gpu<<<GridDim(n), BlockDim, 0, streams[0]>>>(device_cnt, g_devices, g_isdevice);
        qx = (double*)malloc(n * sizeof(double));
        qy = (double*)malloc(n * sizeof(double));
        qz = (double*)malloc(n * sizeof(double));

        for (int step = 0; step <= param::n_steps; step++) {
            double dx, qx_p, qx_a;
            double dy, qy_p, qy_a;
            double dz, qz_p, qz_a;
            if (step == 0) {
                qx_p = qx0[planet];
                qy_p = qy0[planet];
                qz_p = qz0[planet];
                qx_a = qx0[asteroid];
                qy_a = qy0[asteroid];
                qz_a = qz0[asteroid];
            } else {
                hipStreamSynchronize(streams[1]);
                if (thread_id == 0)
                    compute_accelerations_gpu<<<GridDim(n * n), BlockDim, 0, streams[0]>>>(true, step, n, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz, g_ax, g_ay, g_az, g_m, g_isdevice);
                else
                    compute_accelerations_gpu<<<GridDim(n * n), BlockDim, 0, streams[0]>>>(false, step, n, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz, g_ax, g_ay, g_az, g_m, g_isdevice);
                update_velocities_gpu<<<GridDim(3 * n), BlockDim, 0, streams[0]>>>(n, g_vx, g_vy, g_vz, g_ax, g_ay, g_az);
                hipStreamSynchronize(streams[0]);
                clear_a_gpu<<<GridDim(3 * n), BlockDim, 0, streams[1]>>>(n, g_ax, g_ay, g_az);
                update_positions_gpu<<<GridDim(3 * n), BlockDim, 0, streams[0]>>>(n, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz);
                CUDA_CALL(hipMemcpyAsync(qx, g_qx, n * sizeof(double), hipMemcpyDeviceToHost, streams[0]));
                CUDA_CALL(hipMemcpyAsync(qy, g_qy, n * sizeof(double), hipMemcpyDeviceToHost, streams[0]));
                CUDA_CALL(hipMemcpyAsync(qz, g_qz, n * sizeof(double), hipMemcpyDeviceToHost, streams[0]));
                CUDA_CALL(hipStreamSynchronize(streams[0]));
                qx_p = qx[planet];
                qy_p = qy[planet];
                qz_p = qz[planet];
                qx_a = qx[asteroid];
                qy_a = qy[asteroid];
                qz_a = qz[asteroid];
            }
            dx = qx_p - qx_a;
            dy = qy_p - qy_a;
            dz = qz_p - qz_a;
            if (thread_id == 0) {
                min_dist = std::min(min_dist, sqrt(dx * dx + dy * dy + dz * dz));
            } else if (thread_id == 1) {
                if (dx * dx + dy * dy + dz * dz < param::planet_radius * param::planet_radius) {
                    hit_time_step = step;
                    break;
                }
            }
        }
        for (int i = 0; i < param::cuda_nstreams; i++) {
            hipStreamDestroy(streams[i]);
        }
        CUDA_CALL(hipFree(g_qx));
        CUDA_CALL(hipFree(g_qy));
        CUDA_CALL(hipFree(g_qz));
        CUDA_CALL(hipFree(g_vx));
        CUDA_CALL(hipFree(g_vy));
        CUDA_CALL(hipFree(g_vz));
        CUDA_CALL(hipFree(g_ax));
        CUDA_CALL(hipFree(g_ay));
        CUDA_CALL(hipFree(g_az));
        CUDA_CALL(hipFree(g_m));
        CUDA_CALL(hipFree(g_isdevice));
        CUDA_CALL(hipFree(g_devices));
    }

    if (hit_time_step != -2) {
        // Problem 3
        gravity_device_id = -1;
        missile_cost = std::numeric_limits<double>::infinity();
#pragma omp parallel for schedule(static) num_threads(2)
        for (int di = 0; di < device_cnt; di++) {
            int thread_id = omp_get_thread_num();
            hipSetDevice(thread_id);
            hipStream_t streams[param::cuda_nstreams];
            for (int i = 0; i < param::cuda_nstreams; i++) {
                hipStreamCreate(&streams[i]);
            }

            double *qx, *qy, *qz;
            double *g_qx, *g_qy, *g_qz, *g_vx, *g_vy, *g_vz, *g_ax, *g_ay, *g_az, *g_m;
            bool* g_isdevice;
            int* g_devices;

            CUDA_CALL(hipMalloc(&g_qx, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_qy, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_qz, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_vx, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_vy, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_vz, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_ax, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_ay, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_az, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_m, n * sizeof(double)));
            CUDA_CALL(hipMalloc(&g_isdevice, n * sizeof(bool)));
            CUDA_CALL(hipMalloc(&g_devices, device_cnt * sizeof(int)));

            CUDA_CALL(hipMemcpyAsync(g_qx, qx0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_qy, qy0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_qz, qz0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_vx, vx0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_vy, vy0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_vz, vz0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_m, m0, n * sizeof(double), hipMemcpyHostToDevice, streams[0]));
            CUDA_CALL(hipMemcpyAsync(g_devices, devices, device_cnt * sizeof(int), hipMemcpyHostToDevice, streams[0]));
            clear_a_gpu<<<GridDim(3 * n), BlockDim, 0, streams[1]>>>(n, g_ax, g_ay, g_az);
            clear_array_gpu<<<GridDim(n), BlockDim, 0, streams[0]>>>(n, g_isdevice);
            set_isdevice_gpu<<<GridDim(n), BlockDim, 0, streams[0]>>>(device_cnt, g_devices, g_isdevice);
            qx = (double*)malloc(n * sizeof(double));
            qy = (double*)malloc(n * sizeof(double));
            qz = (double*)malloc(n * sizeof(double));

            int d = devices[di];
            bool hit = false;
            bool destroyed = (m0[d] == 0);
            double cost = std::numeric_limits<double>::infinity();

            for (int step = 0; step <= param::n_steps && !hit; step++) {
                double dx, qx_p, qx_a, qx_d;
                double dy, qy_p, qy_a, qy_d;
                double dz, qz_p, qz_a, qz_d;
                if (step == 0) {
                    qx_p = qx0[planet];
                    qy_p = qy0[planet];
                    qz_p = qz0[planet];
                    qx_a = qx0[asteroid];
                    qy_a = qy0[asteroid];
                    qz_a = qz0[asteroid];
                } else {
                    hipStreamSynchronize(streams[1]);
                    compute_accelerations_gpu<<<GridDim(n * n), BlockDim, 0, streams[0]>>>(false, step, n, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz, g_ax, g_ay, g_az, g_m, g_isdevice);
                    update_velocities_gpu<<<GridDim(3 * n), BlockDim, 0, streams[0]>>>(n, g_vx, g_vy, g_vz, g_ax, g_ay, g_az);
                    hipStreamSynchronize(streams[0]);
                    clear_a_gpu<<<GridDim(3 * n), BlockDim, 0, streams[1]>>>(n, g_ax, g_ay, g_az);
                    update_positions_gpu<<<GridDim(3 * n), BlockDim, 0, streams[0]>>>(n, g_qx, g_qy, g_qz, g_vx, g_vy, g_vz);
                    CUDA_CALL(hipMemcpyAsync(qx, g_qx, n * sizeof(double), hipMemcpyDeviceToHost, streams[0]));
                    CUDA_CALL(hipMemcpyAsync(qy, g_qy, n * sizeof(double), hipMemcpyDeviceToHost, streams[0]));
                    CUDA_CALL(hipMemcpyAsync(qz, g_qz, n * sizeof(double), hipMemcpyDeviceToHost, streams[0]));
                    CUDA_CALL(hipStreamSynchronize(streams[0]));
                    qx_p = qx[planet];
                    qy_p = qy[planet];
                    qz_p = qz[planet];
                    qx_a = qx[asteroid];
                    qy_a = qy[asteroid];
                    qz_a = qz[asteroid];
                }
                dx = qx_p - qx_a;
                dy = qy_p - qy_a;
                dz = qz_p - qz_a;
                if (dx * dx + dy * dy + dz * dz < param::planet_radius * param::planet_radius) {
                    hit = true;
                    break;
                }
                if (!destroyed) {
                    if (step == 0) {
                        qx_d = qx0[d];
                        qy_d = qy0[d];
                        qz_d = qz0[d];
                    } else {
                        qx_d = qx[d];
                        qy_d = qy[d];
                        qz_d = qz[d];
                    }
                    dx = qx_p - qx_d;
                    dy = qy_p - qy_d;
                    dz = qz_p - qz_d;
                    double missle_dist = param::missile_speed * step * param::dt;
                    if (dx * dx + dy * dy + dz * dz < missle_dist * missle_dist) {
                        destroyed = true;
                        cost = param::get_missile_cost((step + 1) * param::dt);
                        CUDA_CALL(hipMemsetAsync(g_m + d, 0, sizeof(double), streams[0]));
                    }
                }
            }
#pragma omp critical
            if (!hit && cost < missile_cost) {
                gravity_device_id = d;
                missile_cost = cost;
            }
            for (int i = 0; i < param::cuda_nstreams; i++) {
                hipStreamDestroy(streams[i]);
            }
            CUDA_CALL(hipFree(g_qx));
            CUDA_CALL(hipFree(g_qy));
            CUDA_CALL(hipFree(g_qz));
            CUDA_CALL(hipFree(g_vx));
            CUDA_CALL(hipFree(g_vy));
            CUDA_CALL(hipFree(g_vz));
            CUDA_CALL(hipFree(g_ax));
            CUDA_CALL(hipFree(g_ay));
            CUDA_CALL(hipFree(g_az));
            CUDA_CALL(hipFree(g_m));
            CUDA_CALL(hipFree(g_isdevice));
            CUDA_CALL(hipFree(g_devices));
        }
        if (gravity_device_id == -1) {
            missile_cost = 0;
        }
    }

    write_output(argv[2], min_dist, hit_time_step, gravity_device_id, missile_cost);

    CUDA_CHECK();

    free(qx0);
    free(qy0);
    free(qz0);
    free(vx0);
    free(vy0);
    free(vz0);
    free(m0);
    free(devices);
}

/*
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b20.in outputs/b20.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b30.in outputs/b30.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b40.in outputs/b40.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b50.in outputs/b50.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b60.in outputs/b60.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b70.in outputs/b70.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b80.in outputs/b80.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b90.in outputs/b90.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b100.in outputs/b100.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b200.in outputs/b200.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b512.in outputs/b512.out
make; srun -pipc22 -c2 --gres=gpu:2 ./hw5 testcases/b1024.in outputs/b1024.out

make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b20.in outputs/b20.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b30.in outputs/b30.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b40.in outputs/b40.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b50.in outputs/b50.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b60.in outputs/b60.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b70.in outputs/b70.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b80.in outputs/b80.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b90.in outputs/b90.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b100.in outputs/b100.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b200.in outputs/b200.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b512.in outputs/b512.out
make; srun -pipc22 -c2 --gres=gpu:2 cuda-memcheck ./hw5 testcases/b1024.in outputs/b1024.out

make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b20.in outputs/b20.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b30.in outputs/b30.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b40.in outputs/b40.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b50.in outputs/b50.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b60.in outputs/b60.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b70.in outputs/b70.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b80.in outputs/b80.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b90.in outputs/b90.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b100.in outputs/b100.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b200.in outputs/b200.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b512.in outputs/b512.out
make; srun -pipc22 -c2 --gres=gpu:2 nvprof ./hw5 testcases/b1024.in outputs/b1024.out
 */